
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define BLOCKSIZE 256

__global__ void polynomial_expansion(float *poly, int degree, int n, float *array)
{
    //TODO: Write code to use the GPU here!
    //code should write the output back to array
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
    {
        float temp = array[index];
        float out = 0, xtothepowerof = 1;
        for (int i = 0; i <= degree; i++)
        {
            out += xtothepowerof * poly[i];
            xtothepowerof *= temp;
        }
        array[index] = out;
    }
}

int main(int argc, char *argv[])
{
    //TODO: add usage

    if (argc < 3)
    {
        std::cerr << "usage: " << argv[0] << " n degree" << std::endl;
        return -1;
    }

    int n = atoi(argv[1]); //TODO: atoi is an unsafe function
    int degree = atoi(argv[2]);
    int nbiter = 1;

    float *array = new float[n];
    float *poly = new float[degree + 1];
    for (int i = 0; i < n; ++i)
        array[i] = 1.;

    for (int i = 0; i < degree + 1; ++i)
        poly[i] = 1.;

    float *d_array, *d_poly;

    hipMalloc((void **)&d_array, n * sizeof(float));
    hipMalloc((void **)&d_poly, (degree + 1) * sizeof(float));

    hipMemcpy(d_array, array, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_poly, poly, (degree + 1) * sizeof(float), hipMemcpyHostToDevice);

    std::chrono::time_point<std::chrono::system_clock> begin, end;
    begin = std::chrono::system_clock::now();

    for (int iter = 0; iter < nbiter; ++iter)
        polynomial_expansion<<<(n + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_poly, degree, n, d_array);

    hipDeviceSynchronize();
    end = std::chrono::system_clock::now();
    std::chrono::duration<double> totaltime = (end - begin);
    hipMemcpy(array, d_array, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_array);
    hipFree(d_poly);

    std::cerr << array[0] << std::endl;
    std::cout << n*sizeof(float)/1000 << " " << totaltime.count() << " " << (3*(n)*(degree+1)*nbiter)/(totaltime.count()*1000*1000*1000) << std::endl;

    delete[] array;
    delete[] poly;

    return 0;
}