
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

int main(int argc, char *argv[])
{

    if (argc < 3)
    {
        std::cerr << "usage: " << argv[0] << " n degree" << std::endl;
        return -1;
    }

    int n = atoi(argv[1]); //TODO: atoi is an unsafe function
    int degree = atoi(argv[2]);
    int nbiter = atoi(argv[3]);

    float *array = new float[n];
    float *poly = new float[degree + 1];
    for (int i = 0; i < n; ++i)
        array[i] = 1.;

    for (int i = 0; i < degree + 1; ++i)
        poly[i] = 1.;

    float *d_array, *d_poly;

    hipMalloc((void **)&d_array, n * sizeof(float));
    hipMalloc((void **)&d_poly, (degree + 1) * sizeof(float));

    std::chrono::time_point<std::chrono::system_clock> begin, end;
    begin = std::chrono::system_clock::now();

    for (int iter = 0; iter < nbiter; ++iter)
        hipMemcpy(d_array, array, n * sizeof(float), hipMemcpyHostToDevice);

    end = std::chrono::system_clock::now();
    std::chrono::duration<double> totaltime = (end - begin);
    /*cudaMemcpy(d_poly, poly, (degree + 1) * sizeof(float), cudaMemcpyHostToDevice);



    for (int iter = 0; iter < nbiter; ++iter)
        polynomial_expansion<<<(n + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_poly, degree, n, d_array);

    cudaDeviceSynchronize();
    cudaMemcpy(array, d_array, n * sizeof(float), cudaMemcpyDeviceToHost);
    */
    hipFree(d_array);
    hipFree(d_poly);
    /*end = std::chrono::system_clock::now();
    std::chrono::duration<double> totaltime = (end - begin) / nbiter;

    {
        bool check = true;
        int index;
        for (int i = 0; i < n; ++i)
        {
            if (fabs(array[i] - (degree + 1)) > 0.01)
            {
                check = false;
                index = i;
				break;
            }
        }
        if (!check)
            std::cerr << "Polynomial expansion has corrupted at array[" << index << "]" << std::endl;
    }

    std::cerr << array[0] << std::endl;
    std::cout << n << " " << degree << " " << totaltime.count() << std::endl;*/

    std::cout << "Latency of "<< nbiter <<"times is" << totaltime.count() << std::endl;
    std::cout << "Latency of PCI express is" << totaltime.count() / nbiter << std::endl;

    delete[] array;
    delete[] poly;

    return 0;
}