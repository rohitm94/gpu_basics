#include <iostream>
#include <chrono>

int main(int argc, char *argv[])
{

    if (argc < 3)
    {
        std::cerr << "usage: " << argv[0] << " n degree" << std::endl;
        return -1;
    }

    int n = atoi(argv[1]); //TODO: atoi is an unsafe function
    int nbiter = atoi(argv[2]);

    float *array = new float[n];
    float *poly = new float[degree + 1];
    for (int i = 0; i < n; ++i)
        array[i] = 1.;

    float *d_array;

    hipMalloc((void **)&d_array, n * sizeof(float));

    std::chrono::time_point<std::chrono::system_clock> begin, end;
    begin = std::chrono::system_clock::now();

    for (int iter = 0; iter < nbiter; ++iter)
        hipMemcpy(d_array, array, n * sizeof(float), hipMemcpyHostToDevice);

    end = std::chrono::system_clock::now();
    std::chrono::duration<double> totaltime = (end - begin);

    hipFree(d_array);

    std::cout << "Latency of "<< nbiter <<"times is" << totaltime.count() << std::endl;
    std::cout << "Latency of PCI express is" << totaltime.count() / nbiter << std::endl;

    delete[] array;
    delete[] poly;

    return 0;
}