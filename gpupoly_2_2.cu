
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

int main(int argc, char *argv[])
{


    int n = atoi(argv[1]); //TODO: atoi is an unsafe function
    int nbiter = atoi(argv[2]);

    float *array = new float[n];
    for(int i = 0; i < n; ++i)
        array[i] = 1.;
    float *d_array;

    hipMalloc((void **)&d_array, n * sizeof(float));

    std::chrono::time_point<std::chrono::system_clock> begin, end;
    begin = std::chrono::system_clock::now();

    for(int iter = 0; iter < nbiter; ++iter)
        hipMemcpy(d_array, array, n * sizeof(float), hipMemcpyHostToDevice);

    end = std::chrono::system_clock::now();
    std::chrono::duration<double> totaltime = (end - begin);

    hipFree(d_array);

    std::cout << n*sizeof(float)/1000 <<" "<< (n*sizeof(float))/(totaltime.count()*nbiter) << std::endl;

    delete[] array;

    return 0;
}
