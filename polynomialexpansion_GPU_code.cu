
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define BLOCKSIZE 256

__global__ void polynomial_expansion(float *poly, int degree, int n, float *array)
{
    //TODO: Write code to use the GPU here!
    //code should write the output back to array
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
    {
        float temp = array[index];
        float out = 0, xtothepowerof = 1;
        for (int i = 0; i <= degree; i++)
        {
            out += xtothepowerof * poly[i];
            xtothepowerof *= temp;
        }
        array[index] = out;
    }
}

int main(int argc, char *argv[])
{
    //TODO: add usage

    if (argc < 3)
    {
        std::cerr << "usage: " << argv[0] << " n degree" << std::endl;
        return -1;
    }

    int n = atoi(argv[1]); //TODO: atoi is an unsafe function
    int degree = atoi(argv[2]);
    int nbiter = 1;

    float *array = new float[n];
    float *poly = new float[degree + 1];
    for (int i = 0; i < n; ++i)
        array[i] = 1.;

    for (int i = 0; i < degree + 1; ++i)
        poly[i] = 1.;

    float *d_array, *d_poly;

    hipMalloc((void **)&d_array, n * sizeof(float));
    hipMalloc((void **)&d_poly, (degree + 1) * sizeof(float));

    hipMemcpy(d_array, array, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_poly, poly, (degree + 1) * sizeof(float), hipMemcpyHostToDevice);

    std::chrono::time_point<std::chrono::system_clock> begin, end;
    begin = std::chrono::system_clock::now();

    for (int iter = 0; iter < nbiter; ++iter)
        polynomial_expansion<<<(n + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_poly, degree, n, d_array);

    hipDeviceSynchronize();
    hipMemcpy(array, d_array, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_array);
    hipFree(d_poly);
    end = std::chrono::system_clock::now();
    std::chrono::duration<double> totaltime = (end - begin) / nbiter;

    {
        bool check = true;
        int index;
        for (int i = 0; i < n; ++i)
        {
            if (fabs(array[i] - (degree + 1)) > 0.01)
            {
                check = false;
                index = i;
				break;
            }
        }
        if (!check)
            std::cerr << "Polynomial expansion has corrupted at array[" << index << "]" << std::endl;
    }

    std::cerr << array[0] << std::endl;
    std::cout << n << " " << degree << " " << totaltime.count() << std::endl;

    delete[] array;
    delete[] poly;

    return 0;
}